#include "hip/hip_runtime.h"
#include "util.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void testKernel(int val) {

  val *= 20;
  printf("vall=%d\n", val);
  test_u();
}

int main(int argc, char **argv) {
  hipSetDevice(0); // Ensure device is set

  printf("printf() is called. Output:\n\n");

  dim3 dimGrid(1, 1);
  dim3 dimBlock(1, 1, 1);
  testKernel<<<dimGrid, dimBlock>>>(10);
  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    return EXIT_FAILURE;
  }

  return EXIT_SUCCESS;
}
